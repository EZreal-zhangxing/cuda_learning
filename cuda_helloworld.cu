#include<stdio.h>
#include<iostream>
using namespace std;
#include<hip/hip_runtime.h>

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if(error != hipSuccess){\
        printf("Error: %s:%d ,",__FILE__,__LINE__);\
        printf("Code: %d, reason: %s \n",error,hipGetErrorString(error));\
        exit(-10*error);\
    }\
}

__global__ void helloFromGpu(void){
    printf("hello world from gpus! and block Id_x is  %d thread Id_x is %d \n",blockIdx.x,threadIdx.x);
    // cout << "hello world from gpus!" << endl;
}

int main(void){
    int dev =0;
    cout << "hello world from cpu!" << endl;
    hipDeviceProp_t cudeProp;
    CHECK(hipGetDeviceProperties(&cudeProp,dev));
    printf("Using Device %d: %s\n",dev,cudeProp.name);
    CHECK(hipSetDevice(dev));
    helloFromGpu <<<2,10>>>();
    hipError_t error;
    error = hipGetLastError();
    cout << "cuda status is " << hipGetErrorString(error) <<  "["<< (error == hipSuccess) <<"]" << endl;
    error = hipDeviceReset();
    cout << "cuda status is " << hipGetErrorString(error) <<  "["<< (error == hipSuccess) <<"]" << endl;
    return 0;
}