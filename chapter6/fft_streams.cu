#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<sys/time.h>
#include<hipfft/hipfft.h>
#include<unistd.h>
void random_init(hipComplex * matrix ,int size){
    time_t t;
    srand((unsigned int)time(&t));
    for(int i=0;i<size;i++){
        // matrix[i] = (rand() & 0xFFFFF) / 100;
        matrix[i].x = i;
        matrix[i].y = i;
    }
}


int main(int argc,char * argv[]){
    int nstream = 4;
    hipComplex * h_data[nstream],* d_data,* d_output,* h_output;
    int nums = 10,batch = 128;
    if(argc > 1){
        nums = atoi(argv[1]);
    }
    hipHostMalloc((void **)&h_data[0],sizeof(hipComplex) * nums * batch/nstream);
    hipHostMalloc((void **)&h_data[1],sizeof(hipComplex) * nums * batch/nstream);
    hipHostMalloc((void **)&h_data[2],sizeof(hipComplex) * nums * batch/nstream);
    hipHostMalloc((void **)&h_data[3],sizeof(hipComplex) * nums * batch/nstream);
    hipMalloc((void **)&d_output,sizeof(hipComplex) * nums * batch);
    hipMalloc((void **)&d_data,sizeof(hipComplex) * nums * batch);
    h_output = (hipComplex *)malloc(sizeof(hipComplex) * nums * batch);
    random_init(h_data[0],nums * batch/nstream);
    random_init(h_data[1],nums * batch/nstream);
    random_init(h_data[2],nums * batch/nstream);
    random_init(h_data[3],nums * batch/nstream);
    
    
    hipfftHandle handle[nstream];
    hipStream_t streams[nstream];
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    for(int i=0;i<nstream;i++){
        hipfftCreate(&handle[i]);
        hipStreamCreate(&streams[i]);
    }

    int n[1] = {nums};
    int inembed[2] = {nums,batch/nstream};
    int onembed[2] = {nums,batch/nstream};
    for(int i=0;i<nstream;i++){
        hipfftPlanMany(&handle[i],1,n,inembed,1,nums,onembed,1,nums,HIPFFT_C2C,batch/nstream);
        hipfftSetStream(handle[i],streams[i]);
    }
    hipEventRecord(start);
    for(int i=0;i<nstream;i++){
        hipMemcpyAsync(d_data + i * (batch/nstream) * nums,h_data[i],(batch/nstream) * nums * sizeof(hipComplex),hipMemcpyHostToDevice,streams[i]);
        hipfftExecC2C(handle[i],d_data + i * (batch/nstream) * nums,d_output + i * (batch/nstream) * nums,HIPFFT_FORWARD);
        // hipfftExecC2C(handle[i],h_data + i * (batch/nstream) * nums,d_output + i * (batch/nstream) * nums,HIPFFT_BACKWARD);
        // usleep(2000);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);
    float times = 0;
    hipEventElapsedTime(&times,start,end);
    printf("complex match filter save with rows elapsed time %f us \n",times * 1000.0);
    
    // hipMemcpy(h_output,d_output,sizeof(hipComplex) * nums * batch,hipMemcpyDeviceToHost);
    // for(int i=0;i<batch;i++){
    //     for(int j=0;j<nums;j++){
    //         printf("fft(%5.2f,%5.2f i) = (%5.2f,%5.2f i) ",h_data[i * nums + j].x,h_data[i * nums + j].y,h_output[i * nums + j].x,h_output[i * nums + j].y);
    //     }  
    //     printf("\n");
    // }
    hipHostFree(h_data);
    hipFree(d_output);
    free(h_output);
    hipEventDestroy(start);
    hipEventDestroy(end);
    return 0;
}