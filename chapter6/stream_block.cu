#include<hip/hip_runtime.h>
#include<omp.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<sys/time.h>
#include<hip/device_functions.h>

long seconds(){
    timeval t;
    gettimeofday(&t,NULL);
    return t.tv_sec * 1e6 + t.tv_usec;
}

__global__ void kernal(int sign){
    printf("kernal execute stream id %d\n",sign);
}

__global__ void kernal_1(int sign){
    printf("kernal execute stream id %d\n",sign);
}
__global__ void kernal_2(int sign){
    printf("kernal execute stream id %d\n",sign);
}
__global__ void kernal_3(int sign){
    printf("kernal execute stream id %d\n",sign);
}
__global__ void kernal_4(int sign){
    printf("kernal execute stream id %d\n",sign);
}

void blocking_stream(){
    hipStream_t streams[2];
    hipStreamCreate(streams);
    hipStreamCreate(streams + 1);

    kernal<<<1,1,0,streams[0]>>>(0);
    kernal<<<1,1>>>(-1);
    kernal<<<1,1,0,streams[1]>>>(1);

    printf("host print \n");
    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);

    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);
}

void non_blocking_stream(){
    hipStream_t streams[2];
    hipStreamCreateWithFlags(streams,hipStreamNonBlocking);
    hipStreamCreateWithFlags(streams + 1,hipStreamNonBlocking);

    kernal<<<1,1,0,streams[0]>>>(0);
    kernal<<<1,1>>>(-1);
    kernal<<<1,1,0,streams[1]>>>(1);

    printf("host print \n");
    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);

    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);
}

void event_syn(){
    hipStream_t streams[4];
    hipStreamCreate(streams);
    hipStreamCreate(streams + 1);
    hipStreamCreate(streams + 2);
    hipStreamCreate(streams + 3);

    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for(int i=0;i<4;i++){
        kernal<<<1,1,0,streams[i]>>>(i);
        kernal<<<1,1,0,streams[i]>>>(i);
        // kernal<<<1,1>>>(i);
        kernal<<<1,1,0,streams[i]>>>(i);
        kernal<<<1,1,0,streams[i]>>>(i);
    }
    
    hipEventRecord(end);

    hipEventSynchronize(end);
    float times = 0.0;
    hipEventElapsedTime(&times,start,end);
    printf("all elapsed time %f \n",times);
    for(int i=0;i<4;i++){
        hipStreamDestroy(streams[i]);
    }

}

void deep_first(){
    hipStream_t streams[4];
    hipStreamCreate(streams);
    hipStreamCreate(streams + 1);
    hipStreamCreate(streams + 2);
    hipStreamCreate(streams + 3);

    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for(int i=0;i<4;i++){
        kernal_1<<<1,1,0,streams[i]>>>(i);
        kernal_2<<<1,1,0,streams[i]>>>(i);
        kernal_3<<<1,1,0,streams[i]>>>(i);
        kernal_4<<<1,1,0,streams[i]>>>(i);
    }
    
    hipEventRecord(end);

    hipEventSynchronize(end);
    float times = 0.0;
    hipEventElapsedTime(&times,start,end);
    printf("all elapsed time %f \n",times);
    for(int i=0;i<4;i++){
        hipStreamDestroy(streams[i]);
    }

}

void breadth_first(){
    hipStream_t streams[4];
    hipStreamCreate(streams);
    hipStreamCreate(streams + 1);
    hipStreamCreate(streams + 2);
    hipStreamCreate(streams + 3);

    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for(int i=0;i<4;i++){
        kernal_1<<<1,1,0,streams[i]>>>(i);
    }
    for(int i=0;i<4;i++){
        kernal_2<<<1,1,0,streams[i]>>>(i);
    }
    for(int i=0;i<4;i++){
        kernal_3<<<1,1,0,streams[i]>>>(i);
    }
    for(int i=0;i<4;i++){
        kernal_4<<<1,1,0,streams[i]>>>(i);
    }
    hipEventRecord(end);

    hipEventSynchronize(end);
    float times = 0.0;
    hipEventElapsedTime(&times,start,end);
    printf("all elapsed time %f \n",times);
    for(int i=0;i<4;i++){
        hipStreamDestroy(streams[i]);
    }

}

void CUDART_CB callback(hipStream_t stream,hipError_t status,void * data){
    printf("call back execute \n");
}

void openmp(){
    hipStream_t streams[4];
    hipStreamCreate(streams);
    hipStreamCreate(streams + 1);
    hipStreamCreate(streams + 2);
    hipStreamCreate(streams + 3);

    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    omp_set_num_threads(4);
    #pragma omp parallel
    {   
        int i = omp_get_thread_num();
        kernal_1<<<1,1,0,streams[i]>>>(i);
        kernal_2<<<1,1,0,streams[i]>>>(i);
        kernal_3<<<1,1,0,streams[i]>>>(i);
        kernal_4<<<1,1,0,streams[i]>>>(i);
        hipStreamAddCallback(streams[i],callback,0,0);
    }

    hipEventRecord(end);

    hipEventSynchronize(end);
    float times = 0.0;
    hipEventElapsedTime(&times,start,end);
    printf("all elapsed time %f \n",times);
    for(int i=0;i<4;i++){
        hipStreamDestroy(streams[i]);
    }

}

int main(int argv,char * argc[]){

    // blocking_stream();
    // non_blocking_stream();
    // event_syn();
    // deep_first();
    // cudaDeviceSynchronize();
    // breadth_first();
    openmp();
    hipDeviceReset();
    return 0;
}