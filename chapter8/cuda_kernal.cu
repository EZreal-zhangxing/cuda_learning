#include "hip/hip_runtime.h"
#include "cuda_kernal.h"

long seconds(){
    timeval t;
    gettimeofday(&t,NULL);
    return t.tv_sec * 1e6 + t.tv_usec;
}

void complex_fft_test(hipComplex * input,int paral,int input_size){
    hipComplex * d_input;
    hipMalloc(&d_input,sizeof(hipComplex) * input_size * paral);
    hipMemcpy(d_input,input,sizeof(hipComplex) * input_size * paral,hipMemcpyHostToDevice);

    hipComplex * d_output,* h_output;
    h_output = (hipComplex *)malloc(sizeof(hipComplex) * input_size * paral);
    hipMalloc(&d_output,sizeof(hipComplex) * input_size * paral);

    hipEvent_t start,end;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipfftHandle handle;
    hipfftCreate(&handle);

    // 动目标检测的FFT变换
    int c_n[1] = {input_size};
    int c_inembed_n[2] = {input_size,paral};
    int c_onembed_n[2] = {input_size,paral};
    hipfftPlanMany(&handle,1,c_n,
        c_inembed_n,1,input_size,
        c_onembed_n,1,input_size,HIPFFT_C2C,paral);
    
    hipEventRecord(start);
    hipfftExecC2C(handle,d_input,d_output,HIPFFT_FORWARD);
    hipEventRecord(end);

    hipEventSynchronize(end);
    float times;
    hipEventElapsedTime(&times,start,end);
    printf("[%4d,%4d]complex fft elapsed time %f us\n",input_size,paral,times * 1000);

    // hipMemcpy(h_output,d_output,sizeof(hipComplex) * input_size *  paral,hipMemcpyDeviceToHost);
    
    // for(int i=0;i<input_size * paral;i++){
    //     printf("(%f ,%fi) ",h_output[i].x,h_output[i].y);
    // }
    // printf("\n");
    // printf("(%f ,%fi) ",h_output[input_size * paral - 1].x,h_output[input_size * paral - 1].y);
    hipEventDestroy(start);
    hipEventDestroy(end);
    hipFree(d_input);
    hipFree(d_output);
    free(h_output);
    hipfftDestroy(handle);
}
__global__ void test_array(float ** d_input){
    printf("you input : \n");
    for(int i=0;i<9;i++){
        printf("%f ",d_input[0][i]);
    }
    printf("\n end you input : \n");
}

__global__ void change(float ** d_input,float * d_input_all,int input_size,int paral){
    for(int i=0;i<paral;i++){
        d_input[i] = d_input_all + i * input_size;
    }
}

void float_invert(float * input,int input_size,int paral){
    int size = sqrt(input_size);
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // 此处不能直接使用指针数组，这个是分配在主机内存无法在设备上调用，需要手动在设备上进行分配
    float * d_input_all,*d_output_all;
    float ** d_input,**d_output;
    CHECK(hipMalloc(&d_input,sizeof(float*) * paral));
    CHECK(hipMalloc(&d_output,sizeof(float*) * paral));
    CHECK(hipMalloc(&d_input_all,sizeof(float) * input_size * paral));
    CHECK(hipMemcpy(d_input_all,input,sizeof(float) * input_size * paral,hipMemcpyHostToDevice));
    CHECK(hipMalloc(&d_output_all,sizeof(float) * input_size * paral));
    
    // for(int i=0;i<paral;i++){
    //     d_input[i] = d_input_all + i * input_size;
    //     d_output[i] = d_output_all + i * input_size;
    // }
    change<<<1,1>>>(d_input,d_input_all,input_size,paral);
    change<<<1,1>>>(d_output,d_output_all,input_size,paral);
    test_array<<<1,1>>>(d_input);
    hipDeviceSynchronize();
    int * info;
    hipMalloc((void **)&info,sizeof(int) * paral);
    int * h_info;
    h_info = (int *)malloc(sizeof(int) * paral);

    // int * pivotArray;
    // hipMalloc((void **)&pivotArray,sizeof(int) * size * paral);

    long start = seconds();
    // hipblasSgetrfBatched(handle,size,d_input,size,pivotArray,info,paral);
        
    // hipblasSgetriBatched(handle,size,d_input,size,pivotArray,d_output,size,info,paral);

    // hipEventRecord(start);
    
    CHECK_STATUS(cublasSmatinvBatched(handle,size,d_input,size,d_output,size,info,paral));

    float *h_output;
    h_output = (float *) malloc(sizeof(float) * input_size * paral);
    hipMemcpy(h_output,d_output_all,sizeof(float) * input_size * paral,hipMemcpyDeviceToHost);

    for(int x = 0;x<paral;x++){
        for(int i=0;i<size;i++){
            for(int j=0;j<size;j++){
                printf("%f ",h_output[i* size + j]);
            }
            printf("\n");
        }
        printf("\n");
    }
    
   
    printf("[%4d x %4d] x [%3d]complex invert elapsed time %ld us\n",size,size,paral,(seconds() - start));

    free(h_info);
    hipFree(info);
    hipFree(d_input);
    hipFree(d_output);
    hipblasDestroy(handle);
}

int check_info(int * info,int paral){
    for(int i=0;i<paral;i++){
        if(info[i] != 0){
            printf("info calculate failed !\n");
            return -1;
        }
    }
    return 0;
}

void print_result_invert(int paral,hipComplex * h_output,int size){
    for(int i=0;i<paral * size * size;i++){
        if(i % (size * size) == 0){
            printf("the %d array: \n",i / (size * size));
        }
        if(i % (size * size) % size == 0){
            printf("\n");
        }
        printf("(%f,%fi) ",h_output[i].x,h_output[i].y);
    }
    printf("\n");
}

void complex_invert(hipComplex * input,int input_size,int paral){
    int size = sqrt(input_size);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start,end;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipComplex * d_input_data, * d_output_data;
    CHECK(hipMalloc(&d_input_data,sizeof(hipComplex) * input_size * paral));
    CHECK(hipMemcpy(d_input_data,input,sizeof(hipComplex) * input_size * paral,hipMemcpyHostToDevice));

    CHECK(hipMalloc(&d_output_data,sizeof(hipComplex) * input_size * paral));
    CHECK(hipMemcpy(d_output_data,input,sizeof(hipComplex) * input_size * paral,hipMemcpyHostToDevice));

    hipComplex * data_address[paral],*output_data_address[paral];
    for(int i=0;i<paral;i++){
        data_address[i] = d_input_data + i * input_size;
        output_data_address[i] = d_output_data + i * input_size;
    }

    hipComplex ** d_input,** d_output;
    CHECK(hipMalloc(&d_input,sizeof(hipComplex *) * paral));
    CHECK(hipMalloc(&d_output,sizeof(hipComplex *) * paral));

    CHECK(hipMemcpy(d_input,data_address,sizeof(hipComplex *) * paral,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_output,output_data_address,sizeof(hipComplex *) * paral,hipMemcpyHostToDevice));
    // printf("at ehre 154\n");
    // for(int i=0;i<paral;i++){
    //     CHECK_STATUS(hipblasSetMatrix(size,size,sizeof(hipComplex),input + i * input_size,size,d_input[i],size));
    //     CHECK_STATUS(hipblasSetMatrix(size,size,sizeof(hipComplex),input + i * input_size,size,d_output[i],size));
    // }
    // printf("at ehre 159\n");
    int * info,* h_info;
    CHECK(hipMalloc(&info,sizeof(int) * paral));
    h_info = (int *)malloc(sizeof(int) * paral);
    
    hipEventRecord(start);
    CHECK_STATUS(cublasCmatinvBatched(handle,size,d_input,size,d_output,size,info,paral));
    hipEventRecord(end);
    hipEventSynchronize(end);

    float times;
    hipEventElapsedTime(&times,start,end);
    printf("[%4d x %4d] x [%3d]complex invert elapsed time %f us ",size,size,paral,times * 1000);

    hipMemcpy(h_info,info,sizeof(int) * paral,hipMemcpyDeviceToHost);
    if(check_info(h_info,paral) >= 0){
        printf("\t calculate success !\n");
    }

    // hipComplex * h_output;
    // h_output =(hipComplex *) malloc(sizeof(hipComplex ) * paral * input_size);

    // hipMemcpy(h_output,d_output_data,sizeof(hipComplex) * input_size * paral,hipMemcpyDeviceToHost);

    // print_result_invert(paral,h_output,size);

    hipFree(info);
    free(h_info);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_input_data);
    hipFree(d_output_data);


    hipEventDestroy(start);
    hipEventDestroy(end);
    // free(h_output);
    hipblasDestroy(handle);
}

void complex_matrix_gemm(hipComplex * matrix_a,hipComplex * matrix_b,hipComplex * matrix_c,int m,int n,int k){
    hipEvent_t start,end;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipComplex *d_matrix_a,*d_matrix_b,*d_matrix_c;

    hipMalloc((void **)&d_matrix_a,sizeof(hipComplex) * m * n);
    hipMalloc((void **)&d_matrix_b,sizeof(hipComplex) * n * k);
    hipMalloc((void **)&d_matrix_c,sizeof(hipComplex) * m * k);

    CHECK_STATUS(hipblasSetMatrix(m,n,sizeof(hipComplex),matrix_a,m,d_matrix_a,m));
    CHECK_STATUS(hipblasSetMatrix(n,k,sizeof(hipComplex),matrix_b,n,d_matrix_b,n));
    hipComplex alpha = make_hipComplex(1,0);
    hipComplex belta = make_hipComplex(0,0);
    hipEventRecord(start);
    // CHECK_STATUS(cublasCgemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&alpha,d_matrix_a,HIP_C_32F,m,d_matrix_b,HIP_C_32F,n,&belta,d_matrix_c,HIP_C_32F,m));
    // CHECK_STATUS(cublasCgemm3m(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&alpha,d_matrix_a,m,d_matrix_b,n,&belta,d_matrix_c,m));
    CHECK_STATUS(hipblasCgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&alpha,d_matrix_a,m,d_matrix_b,n,&belta,d_matrix_c,m));
    hipEventRecord(end);

    hipEventSynchronize(end);
    float times;
    hipEventElapsedTime(&times,start,end);
    printf("[%4d,%4d] x [%4d,%4d] complex elapsed time %f us\n",m,n,n,k,times * 1000);

    // hipMemcpy(matrix_c,d_matrix_c,sizeof(hipComplex)*m*k,hipMemcpyDeviceToHost);
    // CHECK_STATUS(hipblasGetMatrix(m,k,sizeof(hipComplex),d_matrix_c,m,matrix_c,m));
    hipFree(d_matrix_a);
    hipFree(d_matrix_b);
    hipFree(d_matrix_c);


    hipEventDestroy(start);
    hipEventDestroy(end);
    hipblasDestroy(handle);

}

void complex_matrix_transpose(hipComplex * input,int width,int height){

    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipComplex * d_input,* d_output;
    CHECK(hipMalloc((void **)&d_input,sizeof(hipComplex) * width * height));
    CHECK(hipMalloc((void **)&d_output,sizeof(hipComplex) * width * height));

    CHECK(hipMemcpy(d_input,input,sizeof(hipComplex) * width * height,hipMemcpyHostToDevice));
    // hipComplex ** h_input_address,*h_output_data,**h_output_address;
    // h_input_address = (hipComplex **)malloc(sizeof(hipComplex *) * input_size);
    // h_output_address = (hipComplex **)malloc(sizeof(hipComplex *) * input_size);
    // h_output_data =(hipComplex *) malloc(sizeof(hipComplex) * input_size);
    // memset(h_output_data,0,sizeof(hipComplex) * input_size);

    // for(int i=0;i<size;i++){
    //     h_input_address[i] = input + size * i;
    //     h_output_address[i] = h_output_data + size * i;
    // }

    // hipComplex * d_input_data,** d_input_address;
    // hipComplex * d_output_data,** d_output_address;

    // CHECK(hipMalloc((void **)&d_input_data,sizeof(hipComplex) * input_size));
    // CHECK(hipMalloc((void **)&d_input_address,sizeof(hipComplex *) * input_size));

    // CHECK(hipMalloc((void **)&d_output_data,sizeof(hipComplex) * input_size));
    // CHECK(hipMalloc((void **)&d_output_address,sizeof(hipComplex *) * input_size));

    // CHECK(hipMemcpy(d_input_data,input,sizeof(hipComplex) * input_size,hipMemcpyHostToDevice));
    // CHECK(hipMemcpy(d_input_address,h_input_address,sizeof(hipComplex *) * size,hipMemcpyHostToDevice));

    // CHECK(hipMemcpy(d_output_data,h_output_data,sizeof(hipComplex) * input_size,hipMemcpyHostToDevice));
    // CHECK(hipMemcpy(d_output_address,h_output_address,sizeof(hipComplex *) * size,hipMemcpyHostToDevice));

    hipComplex alpha = make_hipComplex(1,0);
    hipComplex belta = make_hipComplex(0,0);

    hipEventRecord(start);
    // (h x w)T  + w x h = w x h
    CHECK_STATUS(hipblasCgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,height,width,&alpha,d_input,width,&belta,d_input,height,d_output,height));
    hipEventRecord(end);

    hipEventSynchronize(end);
    float times;
    hipEventElapsedTime(&times,start,end);
    printf("[%5d x %5d]complex transpose elapsed time %f us\n",height,width,times * 1000);

    // hipComplex * h_output;
    // h_output = (hipComplex *)malloc(sizeof(hipComplex) * width * height);
    // hipMemcpy(h_output,d_output,sizeof(hipComplex) * width * height,hipMemcpyDeviceToHost);
    // for(int i=0;i<width;i++){
    //     for(int j=0;j<height;j++){
    //         printf("(%4.2f,%4.2fi) ",h_output[i * height + j].x,h_output[i * height + j].y);
    //     }
    //     printf("\n");
    // }
    
    // free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(end);
    hipblasDestroy(handle);   
}

__global__ void multi_kernal(hipComplex * signal,hipComplex * coefficient,hipComplex * output,int width,int height){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    if(idx < width && idy < height){
        int target_id = idx + idy * width;
        output[target_id].x = signal[target_id].x * coefficient[idx].x - signal[target_id].y * coefficient[idx].y;
        output[target_id].y = signal[target_id].x * coefficient[idx].y + signal[target_id].y * coefficient[idx].x;
    }
}

__global__ void multi_kernal_col(hipComplex * signal,hipComplex * coefficient,hipComplex * output,int width,int height){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    if(idx < width && idy < height){
        int target_id = idx + idy * width;
        output[target_id].x = signal[target_id].x * coefficient[idy].x - signal[target_id].y * coefficient[idy].y;
        output[target_id].y = signal[target_id].x * coefficient[idy].y + signal[target_id].y * coefficient[idy].x;
    }
}
__global__ void multi_kernal_4(hipComplex * signal,hipComplex * coefficient,hipComplex * output,int width,int height){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int expend_idx = idx * 4;
    if(idx < width && idy < height){
        int target_id = idx * 4 + idy * width;
        output[target_id].x = signal[target_id].x * coefficient[expend_idx].x - signal[target_id].y * coefficient[expend_idx].y;
        output[target_id].y = signal[target_id].x * coefficient[expend_idx].y + signal[target_id].y * coefficient[expend_idx].x;

        output[target_id + 1].x = signal[target_id + 1].x * coefficient[expend_idx + 1].x - signal[target_id + 1].y * coefficient[expend_idx + 1].y;
        output[target_id + 1].y = signal[target_id + 1].x * coefficient[expend_idx + 1].y + signal[target_id + 1].y * coefficient[expend_idx + 1].x;

        output[target_id + 2].x = signal[target_id + 2].x * coefficient[expend_idx + 2].x - signal[target_id + 2].y * coefficient[expend_idx + 2].y;
        output[target_id + 2].y = signal[target_id + 2].x * coefficient[expend_idx + 2].y + signal[target_id + 2].y * coefficient[expend_idx + 2].x;

        output[target_id + 3].x = signal[target_id + 3].x * coefficient[expend_idx + 3].x - signal[target_id + 3].y * coefficient[expend_idx + 3].y;
        output[target_id + 3].y = signal[target_id + 3].x * coefficient[expend_idx + 3].y + signal[target_id + 3].y * coefficient[expend_idx + 3].x;
    }
}

/**
 * ifft(fft(sginal) * fft(coefficient))
 * row * col
*/
float match_filter(hipComplex * input_signal,hipComplex * coefficient,int input_size,int paral,int dimx,int dimy){
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    float times = 0.0;
    hipComplex * d_input_signal,* d_coefficient,* d_output;
    size_t data_bytes = sizeof(hipComplex) * input_size * paral,coefficient_bytes = sizeof(hipComplex) * input_size * paral;
    CHECK(hipMalloc((void **)&d_input_signal,data_bytes));
    CHECK(hipMalloc((void **)&d_output,data_bytes));
    CHECK(hipMalloc((void **)&d_coefficient,coefficient_bytes));

    CHECK(hipMemcpy(d_input_signal,input_signal,data_bytes,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_coefficient,coefficient,data_bytes,hipMemcpyHostToDevice));
    hipfftHandle handle_signal,hanlde_coefficient;
    hipfftCreate(&handle_signal);
    hipfftCreate(&hanlde_coefficient);


    /************fft(coefficient)*************/
    int n_c[1] = {input_size};
    int inembed_c[2] = {input_size,paral};
    int onembed_c[2] = {input_size,paral};
    hipfftPlanMany(&hanlde_coefficient,1,n_c,inembed_c,1,input_size,onembed_c,1,input_size,HIPFFT_C2C,paral);

    
    /********************end******************/
    /************fft(signal)*************/
    int n[1] = {input_size};
    int inembed[2] = {input_size,paral};
    int onembed[2] = {input_size,paral};
    hipfftPlanMany(&handle_signal,1,n,inembed,1,input_size,onembed,1,input_size,HIPFFT_C2C,paral);
    /**************end******************/

    dim3 block(dimx,dimy),grid((input_size + block.x - 1)/block.x,(paral + block.y -1)/block.y);
    // block.x /= 4;

    hipEventRecord(start);
    hipfftExecC2C(hanlde_coefficient,d_coefficient,d_coefficient,HIPFFT_FORWARD); // fft(co)
    hipfftExecC2C(handle_signal,d_input_signal,d_input_signal,HIPFFT_FORWARD); // fft(signal)
    multi_kernal<<<grid,block>>>(d_input_signal,d_coefficient,d_output,input_size,paral); // fft(co) * fft(signal)
    
    // multi_kernal_4<<<grid,block>>>(d_input_signal,d_coefficient,d_output,input_size,paral); // fft(co) * fft(signal)
    hipfftExecC2C(handle_signal,d_output,d_output,HIPFFT_BACKWARD); // ifft(fft(co) * fft(signal))
    hipEventRecord(end);

    hipEventSynchronize(end);

    hipEventElapsedTime(&times,start,end);
    printf("[%5d x %5d]complex match filter save with rows elapsed time %f us ",input_size,paral,times * 1000);
    printf("\t multi kernal block:[%d,%d,%d],grid:[%d,%d,%d] \n",block.x,block.y,block.z,grid.x,grid.y,grid.z);

    hipComplex * h_output;
    h_output = (hipComplex *)malloc(data_bytes);
    hipMemcpy(h_output,d_output,data_bytes,hipMemcpyDeviceToHost);

    for(int i=0;i<input_size * paral;i++){
        printf("ifft(fft(%5.2f,%5.2fi) x fft(%5.2f,%5.2fi)) = (%5.2f,%5.2fi) \n",input_signal[i].x,input_signal[i].y,coefficient[i].x,coefficient[i].y,h_output[i].x,h_output[i].y);
    }
    
    free(h_output);
    hipFree(d_coefficient);
    hipFree(d_input_signal);
    hipfftDestroy(handle_signal);
    hipfftDestroy(hanlde_coefficient);
    hipEventDestroy(start);
    hipEventDestroy(end);
    // hipblasDestroy(blas_handle);
    return times * 1000;
}

float match_filter_rowfft_colkernal(hipComplex * input_signal,hipComplex * coefficient,int input_size,int paral,int dimx,int dimy){
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    float times = 0.0;
    hipComplex * d_input_signal,* d_coefficient,* d_output;
    size_t data_bytes = sizeof(hipComplex) * input_size * paral,coefficient_bytes = sizeof(hipComplex) * input_size * paral;
    CHECK(hipMalloc((void **)&d_input_signal,data_bytes));
    CHECK(hipMalloc((void **)&d_output,data_bytes));
    CHECK(hipMalloc((void **)&d_coefficient,coefficient_bytes));

    CHECK(hipMemcpy(d_input_signal,input_signal,data_bytes,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_coefficient,coefficient,data_bytes,hipMemcpyHostToDevice));
    hipfftHandle handle_signal,hanlde_coefficient;
    hipfftCreate(&handle_signal);
    hipfftCreate(&hanlde_coefficient);


    /************fft(coefficient)*************/
    int n_c[1] = {input_size};
    int inembed_c[2] = {input_size,paral};
    int onembed_c[2] = {input_size,paral};
    hipfftPlanMany(&hanlde_coefficient,1,n_c,inembed_c,1,input_size,onembed_c,1,input_size,HIPFFT_C2C,paral);

    
    /********************end******************/
    /************fft(signal)*************/
    int n[1] = {input_size};
    int inembed[2] = {input_size,paral};
    int onembed[2] = {input_size,paral};
    hipfftPlanMany(&handle_signal,1,n,inembed,1,input_size,onembed,1,input_size,HIPFFT_C2C,paral);
    /**************end******************/

    /****************transpose**************/
    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);

    hipComplex alpha = make_hipComplex(1,0);
    hipComplex belta = make_hipComplex(0,0);

    /****************transpose end**************/
    dim3 block(dimx,dimy),grid((input_size + block.x - 1)/block.x,(paral + block.y -1)/block.y);
    // block.x /= 4;

    hipEventRecord(start);
    hipfftExecC2C(hanlde_coefficient,d_coefficient,d_coefficient,HIPFFT_FORWARD); // fft(co)
    hipfftExecC2C(handle_signal,d_input_signal,d_input_signal,HIPFFT_FORWARD); // fft(signal)
    CHECK_STATUS(hipblasCgeam(blas_handle,HIPBLAS_OP_T,HIPBLAS_OP_N,paral,input_size,&alpha,d_input_signal,input_size,&belta,d_input_signal,paral,d_output,paral));
    // multi_kernal<<<grid,block>>>(d_input_signal,d_coefficient,d_output,input_size,paral); // fft(co) * fft(signal)
    multi_kernal_col<<<grid,block>>>(d_output,d_coefficient,d_input_signal,paral,input_size); // fft(co) * fft(signal)
    CHECK_STATUS(hipblasCgeam(blas_handle,HIPBLAS_OP_T,HIPBLAS_OP_N,input_size,paral,&alpha,d_input_signal,paral,&belta,d_input_signal,input_size,d_output,input_size));
    // multi_kernal_4<<<grid,block>>>(d_input_signal,d_coefficient,d_output,input_size,paral); // fft(co) * fft(signal)
    hipfftExecC2C(handle_signal,d_output,d_output,HIPFFT_BACKWARD); // ifft(fft(co) * fft(signal))
    hipEventRecord(end);

    hipEventSynchronize(end);

    hipEventElapsedTime(&times,start,end);
    printf("[%5d x %5d]complex match filter save with rows elapsed time %f us ",input_size,paral,times * 1000);
    printf("\t multi kernal block:[%d,%d,%d],grid:[%d,%d,%d] \n",block.x,block.y,block.z,grid.x,grid.y,grid.z);

    hipComplex * h_output;
    h_output = (hipComplex *)malloc(data_bytes);
    hipMemcpy(h_output,d_output,data_bytes,hipMemcpyDeviceToHost);

    for(int i=0;i<input_size * paral;i++){
        printf("ifft(fft(%5.2f,%5.2fi) x fft(%5.2f,%5.2fi)) = (%5.2f,%5.2fi) \n",input_signal[i].x,input_signal[i].y,coefficient[i].x,coefficient[i].y,h_output[i].x,h_output[i].y);
    }
    
    free(h_output);
    hipFree(d_coefficient);
    hipFree(d_input_signal);
    hipfftDestroy(handle_signal);
    hipfftDestroy(hanlde_coefficient);
    hipEventDestroy(start);
    hipEventDestroy(end);
    hipblasDestroy(blas_handle);
    // hipblasDestroy(blas_handle);
    return times * 1000;
}

float match_filter_col(hipComplex * input_signal,hipComplex * coefficient,int input_size,int paral,int dimx,int dimy){
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    float times = 0.0;
    hipComplex * d_input_signal,* d_coefficient,* d_output;
    size_t data_bytes = sizeof(hipComplex) * input_size * paral,coefficient_bytes = sizeof(hipComplex) * input_size * paral;
    CHECK(hipMalloc((void **)&d_input_signal,data_bytes));
    CHECK(hipMalloc((void **)&d_output,data_bytes));
    CHECK(hipMalloc((void **)&d_coefficient,coefficient_bytes));

    CHECK(hipMemcpy(d_input_signal,input_signal,data_bytes,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_coefficient,coefficient,data_bytes,hipMemcpyHostToDevice));
    hipfftHandle handle_signal,hanlde_coefficient;
    hipfftCreate(&handle_signal);
    hipfftCreate(&hanlde_coefficient);


    /************fft(coefficient)*************/
    int n_c[1] = {input_size};
    int inembed_c[2] = {input_size,paral};
    int onembed_c[2] = {input_size,paral};
    hipfftPlanMany(&hanlde_coefficient,1,n_c,inembed_c,1,input_size,onembed_c,1,input_size,HIPFFT_C2C,paral);
    
    /********************end******************/
    /************fft(signal)*************/
    int n[1] = {input_size};
    int inembed[2] = {paral,input_size};
    int onembed[2] = {paral,input_size};
    if(paral == 1){
        hipfftPlanMany(&handle_signal,1,n,inembed,1,input_size,onembed,1,input_size,HIPFFT_C2C,paral);
    }else{
        hipfftPlanMany(&handle_signal,1,n,inembed,paral,1,onembed,paral,1,HIPFFT_C2C,paral);
    }
    
    /**************end******************/

    dim3 block(dimx,dimy),grid((input_size + block.x - 1)/block.x,(paral + block.y -1)/block.y);
    

    hipEventRecord(start);
    hipfftExecC2C(hanlde_coefficient,d_coefficient,d_coefficient,HIPFFT_FORWARD); // fft(co)
    hipfftExecC2C(handle_signal,d_input_signal,d_input_signal,HIPFFT_FORWARD); // fft(signal)
    multi_kernal_col<<<grid,block>>>(d_input_signal,d_coefficient,d_output,paral,input_size); // fft(co) * fft(signal)
    // block.x /= 4;
    // multi_kernal_4<<<grid,block>>>(d_input_signal,d_coefficient,d_output,input_size,paral); // fft(co) * fft(signal)
    hipfftExecC2C(handle_signal,d_output,d_output,HIPFFT_BACKWARD); // ifft(fft(co) * fft(signal))
    hipEventRecord(end);

    hipEventSynchronize(end);

    hipEventElapsedTime(&times,start,end);
    printf("[%5d x %5d]complex match filter elapsed time %f us ",input_size,paral,times * 1000);
    printf("\t multi kernal block:[%d,%d,%d],grid:[%d,%d,%d] \n",block.x,block.y,block.z,grid.x,grid.y,grid.z);

    // hipComplex * h_output;
    // h_output = (hipComplex *)malloc(data_bytes);
    // hipMemcpy(h_output,d_output,data_bytes,hipMemcpyDeviceToHost);

    // for(int i=0;i<input_size * paral;i++){
    //     printf("ifft(fft(%5.2f,%5.2fi) x fft(%5.2f,%5.2fi)) = (%5.2f,%5.2fi) \n",input_signal[i].x,input_signal[i].y,coefficient[i / paral].x,coefficient[i / paral].y,h_output[i].x,h_output[i].y);
    // }
    
    // free(h_output);
    hipFree(d_coefficient);
    hipFree(d_input_signal);
    hipfftDestroy(handle_signal);
    hipfftDestroy(hanlde_coefficient);
    hipEventDestroy(start);
    hipEventDestroy(end);
    // hipblasDestroy(blas_handle);
    return times * 1000;
}


/**
 * ifft(fft(sginal) * fft(coefficient))
*/
float match_filter_streams(hipComplex * input_signal,hipComplex * coefficient,int input_size,int paral,int dimx,int dimy){
    hipStream_t streams[3];
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);
    hipStreamCreate(&streams[2]);

    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    float times = 0.0;
    hipComplex * d_input_signal,* d_coefficient,* d_output;
    size_t data_bytes = sizeof(hipComplex) * input_size * paral,coefficient_bytes = sizeof(hipComplex) * input_size * paral;
    CHECK(hipMalloc((void **)&d_input_signal,data_bytes));
    CHECK(hipMalloc((void **)&d_output,data_bytes));
    CHECK(hipMalloc((void **)&d_coefficient,coefficient_bytes));

    CHECK(hipMemcpy(d_input_signal,input_signal,data_bytes,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_coefficient,coefficient,data_bytes,hipMemcpyHostToDevice));
    hipfftHandle handle_signal,hanlde_coefficient;
    hipfftCreate(&handle_signal);
    hipfftCreate(&hanlde_coefficient);
    /************fft(coefficient)*************/
    int n_c[1] = {input_size};
    int inembed_c[2] = {input_size,paral};
    int onembed_c[2] = {input_size,paral};
    hipfftPlanMany(&hanlde_coefficient,1,n_c,inembed_c,1,input_size,onembed_c,1,input_size,HIPFFT_C2C,paral);
    hipfftSetStream(hanlde_coefficient,streams[0]);
    /********************end******************/
    /************fft(signal)*************/
    int n[1] = {input_size};
    int inembed[2] = {input_size,paral};
    int onembed[2] = {input_size,paral};
    hipfftPlanMany(&handle_signal,1,n,inembed,1,input_size,onembed,1,input_size,HIPFFT_C2C,paral);
    hipfftSetStream(handle_signal,streams[1]);
    /**************end******************/
    /*****************multi-mv***************/
    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    hipComplex alpha = make_hipComplex(1,0);
    hipComplex beta = make_hipComplex(0,0);
    /********************end*****************/

    dim3 block(dimx,dimy),grid((input_size + block.x - 1)/block.x,(paral + block.y -1)/block.y);

    
    hipEventRecord(start);
    hipfftExecC2C(hanlde_coefficient,d_coefficient,d_coefficient,HIPFFT_FORWARD); // fft(co)
    hipfftExecC2C(handle_signal,d_input_signal,d_input_signal,HIPFFT_FORWARD); // fft(signal)
    // CHECK_STATUS(hipblasCgemv(blas_handle,HIPBLAS_OP_N,paral,input_size,&alpha,d_input_signal,input_size,d_coefficient,1,&beta,d_output,input_size));
    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);
    multi_kernal<<<grid,block,0,streams[2]>>>(d_input_signal,d_coefficient,d_output,input_size,paral); // fft(co) * fft(signal)
    // hipDeviceSynchronize();
    hipStreamSynchronize(streams[2]);
    hipfftExecC2C(handle_signal,d_output,d_output,HIPFFT_BACKWARD); // ifft(fft(co) * fft(signal))

    hipEventRecord(end);

    hipEventSynchronize(end);

    hipEventElapsedTime(&times,start,end);
    printf("[%5d x %5d]complex match filter with 3 streams elapsed time %f us ",input_size,paral,times * 1000);
    printf("\t multi kernal block:[%d,%d,%d],grid:[%d,%d,%d] \n",block.x,block.y,block.z,grid.x,grid.y,grid.z);

    // hipComplex * h_output;
    // h_output = (hipComplex *)malloc(data_bytes);
    // hipMemcpy(h_output,d_output,data_bytes,hipMemcpyDeviceToHost);

    // for(int i=0;i<input_size * paral;i++){
    //     printf("ifft(fft(%5.2f,%5.2fi) x fft(%5.2f,%5.2fi)) = (%5.2f,%5.2fi) \n",input_signal[i].x,input_signal[i].y,coefficient[i].x,coefficient[i].y,h_output[i].x,h_output[i].y);
    // }
    
    // free(h_output);
    hipFree(d_coefficient);
    hipFree(d_input_signal);

    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);
    hipStreamDestroy(streams[2]);
    hipfftDestroy(handle_signal);
    hipfftDestroy(hanlde_coefficient);
    hipEventDestroy(start);
    hipEventDestroy(end);
    // hipblasDestroy(blas_handle);
    return times * 1000;
}

__constant__ hipComplex d_coefficient[8192];

__global__ void conv_kernal(hipComplex * d_input_signal,hipComplex * d_output_signal,int width,int height,int coefficient_size){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int data_idx = idx + idy * width;

    // int data_left_band = idy * width;
    // int data_right_band = (idy + 1) * width;

    int loop = idx > coefficient_size ? coefficient_size:idx;
    hipComplex sum = make_hipComplex(0.0,0.0);
    for(int i=0;i<loop;i++){
        sum.x += d_input_signal[data_idx - i].x * d_coefficient[i].x - d_input_signal[data_idx - i].y * d_coefficient[i].y;
        sum.y += d_input_signal[data_idx - i].x * d_coefficient[i].y + d_input_signal[data_idx - i].y * d_coefficient[i].x;
    }
    d_output_signal[data_idx] = sum;
}

float conv_signal_coeff(hipComplex * input_signal,hipComplex * coefficient,hipComplex * output_signal,int signal_length,int batch,int co_input_size,int dimx,int dimy){
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    float times = 0.0;
    int input_size = signal_length * batch;
    hipComplex * d_input_signal,* d_output_signal;
    
    CHECK(hipMalloc((void **)&d_input_signal,sizeof(hipComplex) * input_size));
    CHECK(hipMalloc((void **)&d_output_signal,sizeof(hipComplex) * input_size));
    // CHECK(hipMalloc((void **)&d_coefficient,sizeof(hipComplex) * co_input_size));

    CHECK(hipMemcpy(d_input_signal,input_signal,sizeof(hipComplex) * input_size,hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_coefficient),coefficient,sizeof(hipComplex) * co_input_size,0,hipMemcpyHostToDevice));
    // CHECK(hipMemcpy(d_coefficient,coefficient,sizeof(hipComplex) * co_input_size,hipMemcpyHostToDevice));
    CHECK(hipMemset(d_output_signal,0,sizeof(hipComplex) * input_size));

    dim3 block(dimx,dimy);
    dim3 grid((signal_length + block.x - 1)/block.x,(batch + block.y - 1) /block.y);
    hipEventRecord(start);
    conv_kernal<<<grid,block>>>(d_input_signal,d_output_signal,signal_length,batch,co_input_size);
    hipEventRecord(end);

    hipEventSynchronize(end);
    hipEventElapsedTime(&times,start,end);
    printf("[%5d x %5d] cov elapsed time %f us ",signal_length,batch,times * 1000);
    printf("\t multi kernal block:[%d,%d,%d],grid:[%d,%d,%d] \n",block.x,block.y,block.z,grid.x,grid.y,grid.z);

    hipMemcpy(output_signal,d_output_signal,sizeof(hipComplex) * input_size,hipMemcpyDeviceToHost);

    hipFree(d_input_signal);
    // hipFree(d_coefficient);
    hipFree(d_output_signal);
    hipEventDestroy(start);
    hipEventDestroy(end);
    return times * 1000;
}

hipStream_t streams[8];
hipEvent_t start,end;

void streamAndEvent_init(){
    for(int i=0;i<8;i++){
        hipStreamCreate(&streams[i]);
    }
    hipEventCreate(&start);
    hipEventCreate(&end);
}

void streamAndEvent_Destroy(){
    for(int i=0;i<8;i++){
        hipStreamDestroy(streams[i]);
    }
    hipEventDestroy(start);
    hipEventDestroy(end);
}
void conv_signal_coeff_async(hipComplex * input_signal,hipComplex * coefficient,hipComplex * output_signal,int signal_length,int batch,int co_input_size,int dimx,int dimy,int streamId){
    
    
    float times = 0.0;
    int input_size = signal_length * batch;
    hipComplex * d_input_signal,* d_coefficient,* d_output_signal;
    
    CHECK(hipMallocAsync((void **)&d_input_signal,sizeof(hipComplex) * input_size,streams[streamId]));
    CHECK(hipMallocAsync((void **)&d_output_signal,sizeof(hipComplex) * input_size,streams[streamId]));
    CHECK(hipMallocAsync((void **)&d_coefficient,sizeof(hipComplex) * co_input_size,streams[streamId]));

    CHECK(hipMemcpyAsync(d_input_signal,input_signal,sizeof(hipComplex) * input_size,hipMemcpyHostToDevice,streams[streamId]));
    CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_coefficient),coefficient,sizeof(hipComplex) * co_input_size,0,hipMemcpyHostToDevice));
    // CHECK(hipMemcpyAsync(d_coefficient,coefficient,sizeof(hipComplex) * co_input_size,hipMemcpyHostToDevice,streams[streamId]));
    // CHECK(hipMemset(d_output_signal,0,sizeof(hipComplex) * input_size));

    dim3 block(dimx,dimy);
    dim3 grid((signal_length + block.x - 1)/block.x,(batch + block.y - 1) /block.y);
    hipEventRecord(start);
    conv_kernal<<<grid,block,0,streams[streamId]>>>(d_input_signal,d_output_signal,signal_length,batch,co_input_size);
    hipEventRecord(end);

    hipEventSynchronize(end);
    hipEventElapsedTime(&times,start,end);
    printf("[%5d x %5d] cov elapsed time %f us ",signal_length,batch,times * 1000);
    printf("\t multi kernal block:[%d,%d,%d],grid:[%d,%d,%d] \n",block.x,block.y,block.z,grid.x,grid.y,grid.z);

    hipMemcpyAsync(output_signal,d_output_signal,sizeof(hipComplex) * input_size,hipMemcpyDeviceToHost,streams[streamId]);

    hipFreeAsync(d_input_signal,streams[streamId]);
    hipFreeAsync(d_coefficient,streams[streamId]);
    hipFreeAsync(d_output_signal,streams[streamId]);

}